#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

#define PCG_DEFAULT_INC_64 1442695040888963407ULL
#define Math_TAU 6.2831853071795864769252867666
#define CMP_EPSILON 0.00001

typedef struct { uint64_t state;  uint64_t inc; } pcg32_random_t;
typedef struct {
    int character;
    int abilityCharacter;
    double abilityLevel;
    int itemCounts[8];
    double startTime;
    int32_t colorState;
    double intensity;
    // would do rgb but cba to figure out imports/packages/whatever or to make my own colour converter
} loadout;

// __device__
// bool seenSeeds[4294967296];

// std::vector<char> characterSet = {
//     '0', '1', '2', '2', '3', '4', '5', '6', '7', '8', '9',
//     'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z'
// };

// helper functions

__forceinline__ __device__
double clamp(double m_a, double m_min, double m_max) {
	if (m_a < m_min) {
		return m_min;
	} else if (m_a > m_max) {
		return m_max;
	}
	return m_a;
}

__forceinline__ __device__
__device__ float lerp(float a, float b, float t) {
    return a + t * (b - a);
}

__device__ void intToChar(uint32_t num, char* str, int maxLength) {
    int index = 0;
    while (num > 0 && index < maxLength - 1) {
        int digit = num % 10;
        str[index++] = '0' + digit;
        num /= 10;
    }
    if (index < maxLength) {
        str[index] = '\0';
    }
}

__forceinline__ __device__
__device__ double pinch(double v) { // function run() uses
	if (v < 0.5) {
		return -v * v;
	}
	return v * v;
}

__device__ double run(double x, double a, double b, double c) { // TorCurve.run() in windowkill
	c = pinch(c);
	x = fmaxf(0, fminf(1, x));
    const float eps = 0.00001f;
	double s = exp(a);
	double s2 = 1.0 / (s + eps);
	double t = fmaxf(0, fminf(1, b));
	double u = c;

	double res, c1, c2, c3;

	if (x < t) {
		c1 = (t * x) / (x + s*(t-x) + eps);
		c2 = t - pow(1/(t+eps), s2-1)*pow(abs(x-t), s2);
		c3 = pow(1/(t+eps), s-1) * pow(x, s);
	} else {
		c1 = (1-t)*(x-1)/(1-x-s*(t-x)+eps) + 1;
		c2 = pow(1/((1-t)+eps), s2-1)*pow(abs(x-t), s2) + t;
		c3 = 1 - pow(1/((1-t)+eps), s-1)*pow(1-x, s);
	}

	if (u <= 0) {
		res = (-u)*c2 + (1+u)*c1;
	} else {
		res = (u)*c3 + (1-u)*c1;
	}

	return res;
}

__forceinline__ __device__
double smoothCorner(double x, double m, double l, double s) { // TorCurve.smoothCorner in windowkill
	double s1 = pow(s/10.0, 2.0);
	return 0.5 * ((l*x + m*(1.0+s1)) - sqrt(pow(abs(l*x-m*(1.0-s1)), 2.0)+4.0*m*m*s1));
}
// end of helper functions

// random number generator (pcg32)

__device__
uint32_t pcg32_random_r(pcg32_random_t* rng);
__device__
void pcg32_srandom_r(pcg32_random_t* rng, uint64_t initstate, uint64_t initseq);
__device__
uint32_t pcg32_boundedrand_r(pcg32_random_t* rng, uint32_t bound);

class RandomPCG {
	pcg32_random_t pcg;
	uint64_t current_seed = 0; // The seed the current generator state started from.
	uint64_t current_inc = 0;

public:
	static const uint64_t DEFAULT_SEED = 12047754176567800795U;
	static const uint64_t DEFAULT_INC = PCG_DEFAULT_INC_64;

    __device__
	RandomPCG(uint64_t p_seed = DEFAULT_SEED, uint64_t p_inc = DEFAULT_INC);

    __device__
	void seed(uint64_t p_seed) {
		current_seed = p_seed;
		pcg32_srandom_r(&pcg, current_seed, current_inc);
	}
    __device__
	uint64_t get_seed() { return current_seed; }

    __device__
	void set_state(uint64_t p_state) { pcg.state = p_state; }
    __device__
	uint64_t get_state() const { return pcg.state; }

    __device__
	uint32_t rand() {
		return pcg32_random_r(&pcg);
	}
    __device__
	uint32_t randbound(uint32_t bounds) {
		return pcg32_boundedrand_r(&pcg, bounds);
	}

    __device__
	double randd() {
		uint32_t proto_exp_offset = rand();
		if (proto_exp_offset == 0) {
			return 0;
		}
		uint64_t significand = (((uint64_t)rand()) << 32) | rand() | 0x8000000000000001U;
		return ldexp((double)significand, -64 - __clzll(proto_exp_offset));
	}
    __device__
	double randf() {
		uint32_t proto_exp_offset = rand();
		if (proto_exp_offset == 0) {
			return 0;
		}
		return (double) (float) (ldexp((double)(rand() | 0x80000001), -32 - __clz(proto_exp_offset)));
	}

    __device__
    double randfn(double p_mean, double p_deviation) {
        double temp = randf();
        if (temp < 0.00001) {
        temp += 0.00001;
    }
        return p_mean + p_deviation * (cos(6.2831853071795864769252867666 * static_cast<double>(randf())) * sqrt(-2.0 * log(static_cast<double>(temp))));
    }

    __device__
	double randomDouble(double p_from, double p_to);
    __device__
	double randomFloat(float p_from, float p_to);
    __device__
	int randomInteger(int p_from, int p_to);
};

__device__
RandomPCG::RandomPCG(uint64_t p_seed, uint64_t p_inc) :
		pcg(),
		current_inc(p_inc) {
	seed(p_seed);
}

__device__
double RandomPCG::randomDouble(double p_from, double p_to) {
	return randd() * (p_to - p_from) + p_from;
}

__device__
double RandomPCG::randomFloat(float p_from, float p_to) {
	return (double) (randf()*(p_to - p_from) + p_from);
}

__device__
int RandomPCG::randomInteger(int p_from, int p_to) {
	if (p_from == p_to) {
		return p_from;
	}
	return randbound(abs(p_from - p_to) + 1) + min(p_from, p_to);
}

__device__
uint32_t pcg32_random_r(pcg32_random_t* rng)
{
    uint64_t oldstate = rng->state;
    
    rng->state = oldstate * 6364136223846793005ULL + (rng->inc|1);
    
    uint32_t xorshifted = ((oldstate >> 18u) ^ oldstate) >> 27u;
    uint32_t rot = oldstate >> 59u;
    return (xorshifted >> rot) | (xorshifted << ((-rot) & 31));
}

__device__
void pcg32_srandom_r(pcg32_random_t* rng, uint64_t initstate, uint64_t initseq)
{
    rng->state = 0U;
    rng->inc = (initseq << 1u) | 1u;
    pcg32_random_r(rng);
    rng->state += initstate;
    pcg32_random_r(rng);
}

__device__
uint32_t pcg32_boundedrand_r(pcg32_random_t *rng, uint32_t bound) {
	uint32_t threshold = -bound % bound;

	for (;;) {
		uint32_t r = pcg32_random_r(rng);
		if (r >= threshold)
			return r % bound;
	}
}

class RandomNumberGenerator {
protected:
	RandomPCG randbase;
public:
    __device__
	void set_seed(uint64_t p_seed) { randbase.seed(p_seed); }
    __device__
	uint64_t get_seed() { return randbase.get_seed(); }

    __device__
	void set_state(uint64_t p_state) { randbase.set_state(p_state); }
    __device__
	uint64_t get_state() const { return randbase.get_state(); }

    __device__
	uint32_t randbound(uint32_t bounds) {
		return randbase.randbound(bounds);
	}
    __device__
	uint32_t randi() { return randbase.rand(); }
    __device__
	double randf() { return randbase.randf(); }
    __device__
	double randf_range(float p_from, float p_to) {
        return randbase.randomFloat(p_from, p_to);
    }
    __device__
	double randfn(float p_mean = 0.0, float p_deviation = 1.0) { return randbase.randfn(p_mean, p_deviation); }
    __device__
	int randi_range(int p_from, int p_to) { return randbase.randomInteger(p_from, p_to); }
    __device__
    void shuffle(int *arr, int n) {
        if (n <= 1) return;

        for (int i = n - 1; i > 0; i--) {
            int j = randbase.randbound(i + 1);
            
            int temp = arr[i];
            arr[i] = arr[j];
            arr[j] = temp;
        }
    }
};

// end of random number generator

// seed function

__device__
loadout get_results(uint64_t seed) {
    RandomNumberGenerator rng;
    RandomNumberGenerator globalRng;


//                                 speed fireRate multiShot wallPunch splashDamage piercing freezing infection
    int itemCategories[8] = {      0,    1,       2,        3,        4,           5,       6,       7        };
    float itemCosts[8] = {         1.0f, 2.8f,    3.3f,     1.25f,    2.0f,        2.4f,    1.5f,    2.15f    };
    //                 basic mage laser melee pointer swarm
    int charList[6] = {0,    1,   2,    3,    4,      5};

    int itemCounts[8];

    rng.set_seed(seed);
    double intensity = rng.randf_range(0.20f, 1.0f);

    int character = charList[rng.randi() % 6];
    int abilityChar = charList[rng.randi() % 6];
    double abilityLevel = 1.0 + round(run(rng.randf(), 1.5/(1.0+intensity),1.0,0.0)*6);

    double itemCount = 8.0;


    double points = 0.66 * itemCount * rng.randf_range(0.5, 1.5) * (1.0 + 4.0*pow(intensity, 1.5));

    double itemDistSteepness = rng.randf_range(-0.5, 2.0);
    
    double itemDistArea = 1.0 / (1.0 + pow(2.0, 0.98*itemDistSteepness));

    globalRng.set_seed(rng.get_seed());
    globalRng.shuffle(itemCategories, 8);
    
    if (rng.randf() < intensity) {
        int multishotIdx = -1;
        for (int i = 0; i < itemCount; ++i) {
            if (itemCategories[i] == 2) {
                multishotIdx = i;
                break;
            }
        }

        if (multishotIdx != -1) {
            // Remove the multishot element
            for (int i = multishotIdx; i < itemCount - 1; ++i) {
                itemCategories[i] = itemCategories[i + 1];
            }
        }

        // Insert multiShot at a new index
        int insertIdx = itemCount - 1 - rng.randi_range(0, 2);
        for (int i = itemCount; i > insertIdx; --i) {
            itemCategories[i] = itemCategories[i - 1];
        }
        itemCategories[insertIdx] = 2;
    }

    if (rng.randf() < intensity) {
        int fireRateIdx = -1;
        for (int i = 0; i < itemCount; ++i) {
            if (itemCategories[i] == 1) {
                fireRateIdx = i;
                break;
            }
        }

        if (fireRateIdx != -1) {
            // Remove the firerate element
            for (int i = fireRateIdx; i < itemCount - 1; ++i) {
                itemCategories[i] = itemCategories[i + 1];
            }
        }

        // Insert firerate at a new index
        int insertIdx = itemCount - 1 - rng.randi_range(0, 2);
        for (int i = itemCount; i > insertIdx; --i) {
            itemCategories[i] = itemCategories[i - 1];
        }
        itemCategories[insertIdx] = 1;
    }

    double catMax = 7.0;
    // int total = 0; // why does this exist?
    for (int i = 0; i < 8; i++) {
        int item = itemCategories[i];
        double catT = (double) i / catMax;
        double cost = itemCosts[item];
        cost = 1.0 + ((cost - 1.0) / 2.5);
        double baseAmount = 0.0;

        double special = 0.0;
        if (i == 7) {
            special += 4.0 * rng.randf_range(0.0, pow(intensity, 2.0));
        }
        double amount = fmax(0.0, 3.0 * run(catT, itemDistSteepness, 1.0, 0.0) + 3.0 * clamp(rng.randfn(0.0, 0.15), -0.5, 0.5));
        
        itemCounts[item] = (int) clamp(round(baseAmount+amount*((points/cost)/(1.0+5.0*itemDistArea))+special), 0.0, 26.0);
    }

    intensity = -0.05 + intensity*lerp(0.33, 1.2, smoothCorner(((double) itemCounts[2]*1.8+(double) itemCounts[1])/12.0, 1.0, 1.0, 4.0)); // TODO: smoothCorner()

    double finalT = rng.randfn((float) pow(intensity, 1.2), 0.05);
    double startTime = clamp(lerp(60.0*2.0, 60.0*20.0, finalT), 60.0*2.0, 60.0*25.0);

    rng.randf();
    rng.randf();
    int colorState = rng.randi_range(0, 2);
    return loadout{character, abilityChar, abilityLevel, {itemCounts[0], itemCounts[1], itemCounts[2], itemCounts[3], itemCounts[4], itemCounts[5], itemCounts[6], itemCounts[7]}, startTime, colorState};
}
__device__ uint32_t djb2Hash(const char *str) {
    unsigned long hash = 5381;
    int c;
    while (c = *str++) {
        hash = ((hash << 5) + hash) + c; /* hash * 33 + c */
    }

    return hash;
}

__device__ bool shouldStop = false;

__device__
uint32_t hash = 0;

__global__
void bruteForce(double timeMin, double timeMax, int colourState, int fireRateMin, int fireRateMax, int freezingMin, int freezingMax, int infectionMin, int infectionMax, int multiShotMin, int multiShotMax, int piercingMin, int piercingMax, int speedMin, int speedMax, int splashDamageMin, int splashDamageMax, int wallPunchMin, int wallPunchMax, int abilityLevelMin, int abilityLevelMax, int character, int ability) {
    
    int totalThreads = blockDim.x * gridDim.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t i = idx;
    loadout loadout = get_results(0);
    for (;i < 4294967296;i = i + totalThreads) {
        loadout = get_results(i);
		if (
            (character == -1 || loadout.character == character) &&
            (ability == -1 || loadout.abilityCharacter == ability) &&
            (colourState == -1 || loadout.colorState == colourState) &&
            loadout.abilityLevel >= abilityLevelMin &&
            loadout.abilityLevel <= abilityLevelMax &&
            loadout.itemCounts[0] >= speedMin &&
            loadout.itemCounts[0] <= speedMax &&
            loadout.itemCounts[1] >= fireRateMin &&
            loadout.itemCounts[1] <= fireRateMax &&
            loadout.itemCounts[2] >= multiShotMin &&
            loadout.itemCounts[2] <= multiShotMax &&
            loadout.itemCounts[3] >= wallPunchMin &&
            loadout.itemCounts[3] <= wallPunchMax &&
            loadout.itemCounts[4] >= splashDamageMin &&
            loadout.itemCounts[4] <= splashDamageMax  &&
            loadout.itemCounts[5] >= piercingMin &&
            loadout.itemCounts[5] <= piercingMax &&
            loadout.itemCounts[6] >= freezingMin &&
            loadout.itemCounts[6] <= freezingMax &&
            loadout.itemCounts[7] >= infectionMin &&
            loadout.itemCounts[7] <= infectionMax &&
            loadout.startTime >= timeMin &&
            loadout.startTime <= timeMax) {
            shouldStop = true;
            hash = i;
        }
        if (shouldStop) {
            return;
        }
    }
}

extern "C" {

__declspec(dllexport) unsigned int startBruteForce(double timeMin, double timeMax, int colourState, int fireRateMin, int fireRateMax, int freezingMin, int freezingMax, int infectionMin, int infectionMax, int multiShotMin, int multiShotMax, int piercingMin, int piercingMax, int speedMin, int speedMax, int splashDamageMin, int splashDamageMax, int wallPunchMin, int wallPunchMax, int abilityLevelMin, int abilityLevelMax, int character, int ability) {
    bruteForce<<<2048,256>>>( timeMin,  timeMax,  colourState,  fireRateMin,  fireRateMax,  freezingMin,  freezingMax,  infectionMin,  infectionMax,  multiShotMin,  multiShotMax,  piercingMin,  piercingMax,  speedMin,  speedMax,  splashDamageMin,  splashDamageMax,  wallPunchMin,  wallPunchMax,  abilityLevelMin,  abilityLevelMax,  character,  ability);

    uint32_t h_hash;

    // Copy winning hash to host memory
    hipMemcpyFromSymbol(&h_hash, HIP_SYMBOL(hash), sizeof(uint32_t), 0, hipMemcpyDeviceToHost);

    return {h_hash};
}
}